
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <string>
#include <png.h>
#include <algorithm>
#include <omp.h>

#define PI std::acos(-1)
#define r 10
#define rs (int)ceil((double)r * 2.57)
#define Rs 53
#define THREAD 256

int read_png(const char *filename, unsigned char **image, unsigned *height, unsigned *width, unsigned *channels);
void write_png(const char *filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels);
void gaussian_filter(double **host_filter_matrix, double *wsum);
__global__ void gaussian_blur(unsigned char *src, unsigned char *tar, double *device_filter_matrix, unsigned start_height, unsigned end_height, unsigned height, unsigned width, unsigned channels, double wsum);

// calculate time
struct timespec start, timeEnd;
double total_time = 0.0;
double timeDiff(struct timespec start, struct timespec timeEnd){
    // function used to measure time in nano resolution
    float output;
    float nano = 1000000000.0;
    if(timeEnd.tv_nsec < start.tv_nsec) output = ((timeEnd.tv_sec - start.tv_sec -1)+(nano+timeEnd.tv_nsec-start.tv_nsec)/nano);
    else output = ((timeEnd.tv_sec - start.tv_sec)+(timeEnd.tv_nsec-start.tv_nsec)/nano);
    return output;
}

int main(int argc, char **argv)
{
    unsigned height, width, channels;
    unsigned char *host_src = NULL;
    unsigned char *host_tar = NULL;
    double *host_filter_matrix = NULL;
    double wsum = 0;

    // cpu number
    cpu_set_t cpu_set;
    sched_getaffinity(0, sizeof(cpu_set), &cpu_set);
    int ncpus = CPU_COUNT(&cpu_set);

    // read image
    if (read_png(argv[1], &host_src, &height, &width, &channels)) {
        std::cout << "[Info]: Cannot read image file \n\n";
        std::cout << "[Info]: Calculation -------- FAIL\n";
        exit(1);
    }

    // allocate host target memory
    host_tar = (unsigned char*)malloc(sizeof(unsigned char) * height * width * channels);

    clock_gettime(CLOCK_MONOTONIC, &start); // get start time

    // precalculate gaussian filter
    gaussian_filter(&host_filter_matrix, &wsum);

    #pragma omp parallel num_threads(ncpus)
    {
        // get thread number
        unsigned cpu_thread_id = omp_get_thread_num();
        //get total thread number
        unsigned cpu_num = omp_get_num_threads();
        // thread set its device
        hipSetDevice(cpu_thread_id);

        unsigned start_height = (height / cpu_num) * cpu_thread_id;
        unsigned end_height = (cpu_thread_id == cpu_num - 1) ? height : start_height + height / cpu_num;

        // allocate device src tar and filter memory
        unsigned char *device_tar = NULL;
        unsigned char *device_src = NULL;
        double *device_filter_matrix = NULL;
        hipMalloc(&device_src, height * (width + THREAD) * channels * sizeof(unsigned char));
        hipMalloc(&device_tar, height * width * channels * sizeof(unsigned char));
        hipMemcpy(device_src, host_src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMalloc(&device_filter_matrix, sizeof(double) * (2 * rs + 1) * (2 * rs + 1));
        hipMemcpy(device_filter_matrix, host_filter_matrix, sizeof(double) * (2 * rs + 1) * (2 * rs + 1), hipMemcpyHostToDevice);


        // calculate block size and thread number
        int thread_num = THREAD;
        int x = (width % thread_num) ? width / thread_num + 1 : width / thread_num;
        int y = 512;
        dim3 blocks(x, y);

        // gaussian blur algorithm
        gaussian_blur<<<blocks, thread_num>>>(device_src, device_tar, device_filter_matrix, start_height, end_height, height, width, channels, wsum);
        

        // write result back to host
        hipMemcpy(&host_tar[start_height * channels * width], &device_tar[start_height * channels * width], (end_height - start_height) * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipFree(device_src);
        hipFree(device_tar);
        hipFree(device_filter_matrix);
        #pragma omp barrier
    }

    clock_gettime(CLOCK_MONOTONIC, &timeEnd); // get end time
    total_time += timeDiff(start, timeEnd); // update computation time

    // write image back
    write_png(argv[2], host_tar, height, width, channels);

    std::cout << "[Info]: Result saved in " << argv[2] << std::endl;
    std::cout << "[Info]: Calculation -------- SUCCESS\n";
    std::cout << "[Info]: Total Executioin time = " << total_time << std::endl;

    // free image array
    free(host_src);
    free(host_tar);
    free(host_filter_matrix);

    return 0;
}

__global__ void gaussian_blur(unsigned char *src, unsigned char *tar, double *device_filter_matrix, unsigned start_height, unsigned end_height, unsigned height, unsigned width, unsigned channels, double wsum) {
    __shared__ unsigned char R_arr[Rs][THREAD + Rs];
    __shared__ unsigned char G_arr[Rs][THREAD + Rs];
    __shared__ unsigned char B_arr[Rs][THREAD + Rs];

    int x, y, iy, a, b, i, j;
    double val[3];
    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockIdx.y + start_height;

    for (; y < end_height; y += gridDim.y) {
        #pragma unroll (5)
        for (iy = y - rs; iy < y + rs + 1; iy++) {
            a = min((int)height-1, max(0, iy));
            R_arr[iy + rs - y][threadIdx.x + rs] = src[channels * (width * a + x) + 2];
            G_arr[iy + rs - y][threadIdx.x + rs] = src[channels * (width * a + x) + 1];
            B_arr[iy + rs - y][threadIdx.x + rs] = src[channels * (width * a + x) + 0];
            if (threadIdx.x < rs) {
                b = min((int)width-1, max(0, x - rs));
                R_arr[iy + rs - y][threadIdx.x] = src[channels * (width * a + b) + 2];
                G_arr[iy + rs - y][threadIdx.x] = src[channels * (width * a + b) + 1];
                B_arr[iy + rs - y][threadIdx.x] = src[channels * (width * a + b) + 0];
            } else if (THREAD - 1 - threadIdx.x < rs) {
                b = min((int)width-1, max(0, x + rs));
                R_arr[iy + rs - y][threadIdx.x + 2 * rs] = src[channels * (width * a + b) + 2];
                G_arr[iy + rs - y][threadIdx.x + 2 * rs] = src[channels * (width * a + b) + 1];
                B_arr[iy + rs - y][threadIdx.x + 2 * rs] = src[channels * (width * a + b) + 0];
            }
        }
        __syncthreads();

        val[0] = val[1] = val[2] = 0.0;
        for (i = 0; i < Rs; i++) {
            for (j = 0; j < Rs; j++) {
                val[2] += (double)R_arr[i][threadIdx.x + j] * device_filter_matrix[Rs * i + j];
                val[1] += (double)G_arr[i][threadIdx.x + j] * device_filter_matrix[Rs * i + j];
                val[0] += (double)B_arr[i][threadIdx.x + j] * device_filter_matrix[Rs * i + j];
            }
        }
        if (x < width) {
            tar[channels * (width * y + x) + 2] = round(val[2] / wsum);
            tar[channels * (width * y + x) + 1] = round(val[1] / wsum);
            tar[channels * (width * y + x) + 0] = round(val[0] / wsum);
        }
    }
}

void gaussian_filter(double **host_filter_matrix, double *wsum) {
    int dsq;
    double a = (double)(PI * 2 * r * r);
    double b = 2 * r * r;
    double wght;
    
    (*host_filter_matrix) = (double*)malloc(sizeof(double) * (2 * rs + 1) * (2 * rs + 1));

    for (int i = 0; i <= 2 * rs; i++) {
        for (int j = 0; j <= 2 * rs; j++) {
            dsq = (i - rs) * (i - rs) + (j - rs) * (j - rs);
            wght = exp((double)(-1 * dsq) / b) / a;
            (*host_filter_matrix)[(2 * rs + 1) * i + j] = wght;
            (*wsum) += wght;
        }
     }
}

int read_png(const char *filename, unsigned char **image, unsigned *height, unsigned *width, unsigned *channels) {
    unsigned char sig[8];
    FILE *infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    // cudaMallocHost(&host_t, height * width * channels * sizeof(unsigned char));
    if ((*image = (unsigned char *)malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0; i < *height; ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char *filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels) {
    FILE *fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}