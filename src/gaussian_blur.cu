
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <string>
#include <png.h>
#include <algorithm>

#define PI std::acos(-1)
#define r 10
#define rs (int)ceil((double)r * 2.57)
#define Rs 53 // filter matrix size
#define BK 16 // block dim
#define THREAD 256

int read_png(const char *filename, unsigned char **image, unsigned *height, unsigned *width, unsigned *channels);
void write_png(const char *filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels);
void gaussian_filter(double **host_filter_matrix, double *wsum);
__global__ void gaussian_blur(unsigned char *src, unsigned char *tar, double *device_filter_matrix, unsigned height, unsigned width, unsigned channels, double wsum);

// calculate time
struct timespec start, timeEnd;
double total_time = 0.0;
double timeDiff(struct timespec start, struct timespec timeEnd){
    // function used to measure time in nano resolution
    float output;
    float nano = 1000000000.0;
    if(timeEnd.tv_nsec < start.tv_nsec) output = ((timeEnd.tv_sec - start.tv_sec -1)+(nano+timeEnd.tv_nsec-start.tv_nsec)/nano);
    else output = ((timeEnd.tv_sec - start.tv_sec)+(timeEnd.tv_nsec-start.tv_nsec)/nano);
    return output;
}

int main(int argc, char **argv)
{
    unsigned height, width, channels;
    unsigned char *host_src = NULL;
    unsigned char *device_src = NULL;
    unsigned char *host_tar = NULL;
    unsigned char *device_tar = NULL;
    double *host_filter_matrix = NULL;
    double *device_filter_matrix = NULL;
    double wsum = 0;

    // read image
    if (read_png(argv[1], &host_src, &height, &width, &channels)) {
        std::cout << "[Info]: Cannot read image file \n\n";
        std::cout << "[Info]: Calculation -------- FAIL\n";
        exit(1);
    }

    // allocate memory
    host_tar = (unsigned char*)malloc(sizeof(unsigned char) * height * width * channels);

    // allocate device_src more memory to prevent out of memory
    clock_gettime(CLOCK_MONOTONIC, &start); // get start time
    hipMalloc(&device_src, height * width * channels * sizeof(unsigned char));
    hipMalloc(&device_tar, height * width * channels * sizeof(unsigned char));
    hipMemcpy(device_src, host_src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // precalculate gaussian filter
    gaussian_filter(&host_filter_matrix, &wsum);

    hipMalloc((void**)(&device_filter_matrix), sizeof(double) * (2 * rs + 1) * (2 * rs + 1));
    hipMemcpy((void*)device_filter_matrix, host_filter_matrix, sizeof(double) * (2 * rs + 1) * (2 * rs + 1), hipMemcpyHostToDevice);

    // calculate block size and thread number
    int x = (width % BK) ? width / BK + 1 : width / BK;
    int y = (height % BK) ? height / BK + 1 : height / BK;
    dim3 blocks_size(x, y);
    dim3 threads_size(THREAD);

    // gaussian blur algorithm
    gaussian_blur<<<blocks_size, threads_size>>>(device_src, device_tar, device_filter_matrix, height, width, channels, wsum);

    clock_gettime(CLOCK_MONOTONIC, &timeEnd); // get end time
    total_time += timeDiff(start, timeEnd); // update computation time

    // write result back to host
    hipMemcpy(host_tar, device_tar, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
    // clock_gettime(CLOCK_MONOTONIC, &timeEnd); // get end time
    // total_time += timeDiff(start, timeEnd); // update computation time

    // write image back
    write_png(argv[2], host_tar, height, width, channels);

    std::cout << "[Info]: Result saved in " << argv[2] << std::endl;
    std::cout << "[Info]: Calculation -------- SUCCESS\n";
    std::cout << "[Info]: Total Executioin time = " << total_time << std::endl;

    // free image array
    free(host_src);
    free(host_tar);
    free(host_filter_matrix);
    hipFree(device_src);
    hipFree(device_tar);
    hipFree(device_filter_matrix);

    return 0;
}

__global__ void gaussian_blur(unsigned char *src, unsigned char *tar, double *device_filter_matrix, unsigned height, unsigned width, unsigned channels, double wsum) {
    __shared__ unsigned char R_arr[Rs + BK][Rs + BK];
    __shared__ unsigned char G_arr[Rs + BK][Rs + BK];
    __shared__ unsigned char B_arr[Rs + BK][Rs + BK];

    int row_block = blockIdx.y * BK;
    int col_block = blockIdx.x * BK;
    int row_pixel = row_block + threadIdx.x / BK;
    int col_pixel = col_block + threadIdx.x % BK;
    int row_inner = threadIdx.x / BK;
    int col_inner = threadIdx.x % BK;
    int get_row_pixel, get_col_pixel;
    double result[3];

    for (int i = threadIdx.x / 64; i < Rs + BK; i += 4) { // 4 = threadnum / 64
        for (int j = threadIdx.x % 64; j < Rs + BK; j += 64) {
            get_row_pixel = min(height - 1, max(0, row_block - rs + i));
            get_col_pixel = min(width - 1, max(0, col_block - rs + j));
            R_arr[i][j] = src[(get_row_pixel * width + get_col_pixel) * channels + 2];
            G_arr[i][j] = src[(get_row_pixel * width + get_col_pixel)* channels + 1];
            B_arr[i][j] = src[(get_row_pixel * width + get_col_pixel) * channels + 0];
        }
    }
    __syncthreads();

    if (row_pixel < height and col_pixel < width) {
        result[0] = result[1] = result[2] = 0.0;
        for (int i = 0; i <= Rs; i++) {
            for (int j = 0; j <= Rs; j++) {
                result[2] += (double)R_arr[row_inner + i][col_inner + j] * device_filter_matrix[Rs * j + i];
                result[1] += (double)G_arr[row_inner + i][col_inner + j] * device_filter_matrix[Rs * j + i];
                result[0] += (double)B_arr[row_inner + i][col_inner + j] * device_filter_matrix[Rs * j + i];
            }
        }

        tar[channels * (row_pixel * width + col_pixel) + 2] = round(result[2] / wsum);
        tar[channels * (row_pixel * width + col_pixel) + 1] = round(result[1] / wsum);
        tar[channels * (row_pixel * width + col_pixel) + 0] = round(result[0] / wsum);
    }
    
}

void gaussian_filter(double **host_filter_matrix, double *wsum) {
    int dsq;
    double a = (double)(PI * 2 * r * r);
    double b = 2 * r * r;
    double wght;
    
    (*host_filter_matrix) = (double*)malloc(sizeof(double) * (2 * rs + 1) * (2 * rs + 1));

    for (int i = 0; i <= 2 * rs; i++) {
        for (int j = 0; j <= 2 * rs; j++) {
            dsq = (i - rs) * (i - rs) + (j - rs) * (j - rs);
            wght = exp((double)(-1 * dsq) / b) / a;
            (*host_filter_matrix)[(2 * rs + 1) * i + j] = wght;
            (*wsum) += wght;
        }
     }
}

int read_png(const char *filename, unsigned char **image, unsigned *height, unsigned *width, unsigned *channels) {
    unsigned char sig[8];
    FILE *infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    // cudaMallocHost(&host_t, height * width * channels * sizeof(unsigned char));
    if ((*image = (unsigned char *)malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0; i < *height; ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char *filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels) {
    FILE *fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}